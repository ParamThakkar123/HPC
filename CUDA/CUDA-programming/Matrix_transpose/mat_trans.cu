#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

#define N 1024  // Matrix size N x N

__global__ void matrixTransposeKernel(float* input, float* output, int n) {
    __shared__ float tile[16][16];  // Shared memory for tiles

    // Global indices for reading from input
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (row < n && col < n) {
        tile[threadIdx.y][threadIdx.x] = input[row * n + col];
    }
    __syncthreads();

    // Global indices for writing to output (transposed)
    row = blockIdx.x * blockDim.y + threadIdx.y;  // Swap blockIdx.x and blockIdx.y
    col = blockIdx.y * blockDim.x + threadIdx.x;

    // Write transposed tile to output
    if (row < n && col < n) {
        output[row * n + col] = tile[threadIdx.x][threadIdx.y];  // Transpose within tile
    }
}

void cpuMatrixTranspose(float* input, float* output, int n) {
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
            output[j * n + i] = input[i * n + j];
}

int main() {
    size_t bytes = N * N * sizeof(float);

    // Allocate pinned memory for host matrices
    float* h_input;
    float* h_output_cpu;
    float* h_output_gpu;
    hipHostMalloc(&h_input, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_output_cpu, bytes, hipHostMallocDefault);
    hipHostMalloc(&h_output_gpu, bytes, hipHostMallocDefault);

    // Initialize input matrix
    for (int i = 0; i < N * N; ++i)
        h_input[i] = static_cast<float>(rand()) / RAND_MAX;

    // CPU Transpose
    auto start_cpu = std::chrono::high_resolution_clock::now();
    cpuMatrixTranspose(h_input, h_output_cpu, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_time = end_cpu - start_cpu;

    // Allocate device memory
    float *d_input, *d_output;
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);

    // Copy input to device
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    // Kernel launch parameters
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // GPU Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrixTransposeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipEventRecord(stop);

    // Copy result back to host
    hipMemcpy(h_output_gpu, d_output, bytes, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, stop);

    // Validate correctness
    bool correct = true;
    for (int i = 0; i < N * N; ++i) {
        if (abs(h_output_cpu[i] - h_output_gpu[i]) > 1e-5) {
            correct = false;
            break;
        }
    }

    // Print results
    std::cout << "Matrix size: " << N << " x " << N << "\n";
    std::cout << "CPU time: " << cpu_time.count() << " ms\n";
    std::cout << "GPU time: " << gpu_time << " ms\n";
    std::cout << "Result: " << (correct ? "Success ✅" : "Mismatch ❌") << "\n";

    // Free memory
    hipHostFree(h_input);
    hipHostFree(h_output_cpu);
    hipHostFree(h_output_gpu);
    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}