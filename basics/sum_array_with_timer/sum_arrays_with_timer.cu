#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>

#define CHECK(call)                                                      \
 {                                                                        \
   const hipError_t error = call;                                       \
   if (error != hipSuccess)                                             \
   {                                                                     \
      printf("Error: %s:%d, ", __FILE__, __LINE__);                      \
      printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
      exit(1);                                                           \
   }                                                                     \
 }


double cpuSecond() {
    using namespace std::chrono;
    return duration_cast<duration<double>>(steady_clock::now().time_since_epoch()).count();
}
void initialData(float *ip, int size) {
    time_t t;
    srand((unsigned int) time(&t));
    for (int i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            return;
        }
    }
    printf("Arrays match.\n");
}

int main(int argc, char **argv){
    printf("%s Starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    int nElem = 1 << 24;
    printf("Vector size %d\n", nElem);

    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    double iStart, iElaps;
    iStart = cpuSecond();

    initialData (h_A, nElem);
    initialData (h_B, nElem);
    iElaps = cpuSecond() - iStart;

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    iStart = cpuSecond();
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    iElaps = cpuSecond() - iStart;

    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    int iLen = 1024;
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);

    iStart = cpuSecond();
    sumArraysOnGPU<<< grid, block  >>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnGPU <<<%d, %d>>> elapsed %f sec\n", grid.x, block.x, iElaps);

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);
    return 0;
}